#include <iostream>   
#include <cstdlib>   
#include <iomanip>
#include <hip/hip_runtime.h>
#include <chrono>
using namespace std::chrono;
using namespace std;

__global__ void vector_add_gpu(float *a, float *b, float *c, int N) {
    int i = threadIdx.x;
    if (i < N) {
        c[i] = a[i] + b[i];
    }
}

void vector_add_cpu(float *a, float *b, float *c, int N) {
    for (int i=0; i<N; i++) {
        if (i < N) {
            c[i] = a[i] + b[i];
        }
    
    }
    cout << "v_c[0] = " << c[0] << ", v_c[1] = " << c[1] << ", v_c[" << N << "] = " << c[N-1] << " ...\n";
    cout << "--------------------------------------------" << endl;

}


int main() {
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    cout << "Using device: "<< prop.name << endl;
    // ---------------------------------------------------------------------------//
    int N = 10;
    float *h_va = (float*)malloc(N * sizeof(float));
    float *h_vb = (float*)malloc(N * sizeof(float));
    float *h_vc = (float*)malloc(N * sizeof(float));
    float *d_va, *d_vb, *d_vc;
    for (int i = 0; i < N; i++) {
        if (i == 0) {
            h_va[i] = 0;
        }
        h_va[i] = i;
        h_vb[i] = (2.0f + i)/3.0f;
        h_vc[i] = 0;
    }
    hipMalloc((void**)&d_va, N*sizeof(float));
    hipMalloc((void**)&d_vb, N*sizeof(float));
    hipMalloc((void**)&d_vc, N*sizeof(float));
    hipMemcpy(d_va, h_va, N*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_vb, h_vb, N*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_vc, h_vc, N*sizeof(float), hipMemcpyHostToDevice);


    dim3 grid_size(1);
    dim3 block_size(N);

    vector_add_cpu(h_va, h_vb, h_vc, N);
  
    hipEvent_t gpu_start, gpu_stop;
    hipEventCreate(&gpu_start);
    hipEventCreate(&gpu_stop);
    hipEventRecord(gpu_start);
    
    vector_add_gpu<<<grid_size, block_size>>>(d_va, d_vb, d_vc, N);

    hipEventRecord(gpu_stop);
    hipEventSynchronize(gpu_stop);

    hipMemcpy(h_vc, d_vc, N*sizeof(float), hipMemcpyDeviceToHost);
    for (int i=0; i < N; i++) {
        cout << "v_c[" << i << "] = " << h_vc[i] << endl;
    }

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, gpu_start, gpu_stop);
    cout << "GPU time: " << milliseconds * 1000 << " us" << endl;

    free(h_va);
    free(h_vb);
    free(h_vc);
    hipFree(d_va); 
    hipFree(d_vb); 
    hipFree(d_vc);

    hipEventDestroy(gpu_start);
    hipEventDestroy(gpu_stop);
    return 0;
} 
