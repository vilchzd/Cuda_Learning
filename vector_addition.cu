#include <iostream>   
#include <cstdlib>   
#include <iomanip>
#include <hip/hip_runtime.h>
#include <chrono>
using namespace std::chrono;
using namespace std;

__global__ void vector_add_gpu(float *a, float *b, float *c, int N) {
    int i = threadIdx.x;
    if (i < N) {
        c[i] = a[i] + b[i];
    }
}

void vector_add_cpu(float *a, float *b, float *c, int N) {
    cout << "--------------------------------------------" << endl;
    for (int i=0; i<N; i++) {
        if (i < N) {
            c[i] = a[i] + b[i];
        }
    
    }
    cout << "v_c[0] = " << c[0] << ", v_c[1] = " << c[1] << ", v_c[" << N-1 << "] = " << c[N-1] << " ...\n";
}


int main() {
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    cout << "Using device: "<< prop.name << endl;
    // ---------------------------------------------------------------------------//
    int N = 512;
    cout << "N= " << N << endl;
    float *h_va = (float*)malloc(N * sizeof(float));
    float *h_vb = (float*)malloc(N * sizeof(float));
    float *h_vc = (float*)malloc(N * sizeof(float));
    float *d_va, *d_vb, *d_vc;
    for (int i = 0; i < N; i++) {
        if (i == 0) {
            h_va[i] = 0;
        }
        h_va[i] = i;
        h_vb[i] = (2.0f + i)/3.0f;
        h_vc[i] = 0;
    }
    hipMalloc((void**)&d_va, N*sizeof(float));
    hipMalloc((void**)&d_vb, N*sizeof(float));
    hipMalloc((void**)&d_vc, N*sizeof(float));
    hipMemcpy(d_va, h_va, N*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_vb, h_vb, N*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_vc, h_vc, N*sizeof(float), hipMemcpyHostToDevice);


    dim3 grid_size(1);
    dim3 block_size(N);

    auto start = high_resolution_clock::now();
    vector_add_cpu(h_va, h_vb, h_vc, N);
    auto stop = high_resolution_clock::now();
    auto duration = duration_cast<microseconds>(stop - start);
    cout << "Time elapsed: " << duration.count() << " us" << endl;
  
    hipEvent_t gpu_start, gpu_stop;
    hipEventCreate(&gpu_start);
    hipEventCreate(&gpu_stop);
    hipEventRecord(gpu_start);
    
    vector_add_gpu<<<grid_size, block_size>>>(d_va, d_vb, d_vc, N);

    hipEventRecord(gpu_stop);
    hipEventSynchronize(gpu_stop);

    hipMemcpy(h_vc, d_vc, N*sizeof(float), hipMemcpyDeviceToHost);
    cout << "--------------------------------------------" << endl;
    cout << "v_c[0] = " << h_vc[0] << ", v_c[1] = " << h_vc[1] << ", v_c[" << N-1 << "] = " << h_vc[N-1] << " ...\n";

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, gpu_start, gpu_stop);
    cout << "GPU time: " << milliseconds * 1000 << " us" << endl;

    free(h_va);
    free(h_vb);
    free(h_vc);
    hipFree(d_va); 
    hipFree(d_vb); 
    hipFree(d_vc);

    hipEventDestroy(gpu_start);
    hipEventDestroy(gpu_stop);
    return 0;
} 
